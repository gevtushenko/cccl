/******************************************************************************
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>

#include <cuda/std/optional>

#include <c2h/catch2_test_helper.h>

C2H_TEST("hipcub::DeviceReduce::Reduce accepts determinism requirements", "[reduce][env]")
{
  // example-begin reduce-env-determinism
  namespace stdexec = cuda::std::execution;

  auto op     = cuda::std::plus{};
  auto input  = c2h::device_vector<float>({0.0f, 1.0f, 2.0f, 3.0f});
  auto output = c2h::device_vector<float>(1);
  auto init   = 0.0f;

  auto requirements = cuda::execution::require(cuda::execution::determinism::run_to_run);

  hipcub::DeviceReduce::Reduce(input.begin(), output.begin(), input.size(), op, init, requirements);

  c2h::device_vector<float> expected{6.0f};
  // example-end reduce-env-determinism

  REQUIRE(output == expected);
}
