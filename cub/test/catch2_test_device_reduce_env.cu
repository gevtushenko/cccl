/******************************************************************************
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Should precede any includes
struct stream_registry_factory_t;
#define CUB_DETAIL_DEFAULT_KERNEL_LAUNCHER stream_registry_factory_t

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include "catch2_test_env_launch_helper.h"

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Reduce, device_reduce);

// %PARAM% TEST_LAUNCH lid 0:1:2

#include <cuda/std/optional>

#include <c2h/catch2_test_helper.h>

namespace stdexec = cuda::std::execution;

// Launcher helper always passes an environment.
// We need a test of simple use to check if default environment works.
// ifdef it out not to spend time compiling and runing it twice.
#if TEST_LAUNCH == 0
struct block_size_check_t
{
  int* ptr;

  __device__ int operator()(int a, int b)
  {
    *ptr = blockDim.x;
    return a + b;
  }
};

struct block_size_retreiver_t
{
  int* ptr;

  template <class ActivePolicyT>
  hipError_t Invoke()
  {
    *ptr = ActivePolicyT::SingleTilePolicy::BLOCK_THREADS;
    return hipSuccess;
  }
};

TEST_CASE("Device reduce works with default environment", "[reduce][device]")
{
  using num_items_t = int;
  using value_t     = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;
  using policy_t    = hipcub::detail::reduce::default_tuning::type<value_t, offset_t, block_size_check_t>::MaxPolicy;

  int current_device{};
  REQUIRE(hipSuccess == hipGetDevice(&current_device));

  int ptx_version{};
  REQUIRE(hipSuccess == hipcub::PtxVersion(ptx_version, current_device));

  int target_block_size{};
  block_size_retreiver_t block_size_retreiver{&target_block_size};
  REQUIRE(hipSuccess == policy_t::Invoke(ptx_version, block_size_retreiver));

  num_items_t num_items = 1;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};
  auto d_in  = thrust::make_constant_iterator(value_t{1});
  auto d_out = thrust::device_vector<value_t>(1);

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Reduce(d_in, d_out.begin(), num_items, block_size_check, value_t{0}));
  REQUIRE(d_out[0] == num_items);

  // Make sure we use default tuning
  REQUIRE(d_block_size[0] == target_block_size);
}

template <int BlockThreads>
struct reduce_tuning : hipcub::detail::reduce::reduce_tuning<reduce_tuning<BlockThreads>>
{
  template <class /* AccumT */, class /* Offset */, class /* OpT */>
  struct type
  {
    struct Policy500 : hipcub::ChainedPolicy<500, Policy500, Policy500>
    {
      struct ReducePolicy
      {
        static constexpr int VECTOR_LOAD_LENGTH = 1;

        static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BLOCK_REDUCE_WARP_REDUCTIONS;

        static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::LOAD_DEFAULT;

        static constexpr int ITEMS_PER_THREAD = 1;
        static constexpr int BLOCK_THREADS    = BlockThreads;
      };

      using SingleTilePolicy      = ReducePolicy;
      using SegmentedReducePolicy = ReducePolicy;
    };

    using MaxPolicy = Policy500;
  };
};

struct get_scan_tuning_query_t
{};

struct scan_tuning
{
  [[nodiscard]] _CCCL_TRIVIAL_API constexpr auto query(const get_scan_tuning_query_t&) const noexcept
  {
    return *this;
  }

  // Make sure this is not used
  template <class /* AccumT */, class /* Offset */, class /* OpT */>
  struct type
  {};
};

using block_sizes = c2h::type_list<cuda::std::integral_constant<int, 32>, cuda::std::integral_constant<int, 64>>;

C2H_TEST("Device reduce can be tuned", "[reduce][device]", block_sizes)
{
  constexpr int target_block_size = c2h::get<0, TestType>::value;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};

  auto num_items = 1;
  auto d_in      = thrust::make_constant_iterator(1);
  auto d_out     = thrust::device_vector<int>(1);

  // We are expecting that `scan_tuning` is ignored
  auto env = cuda::execution::tune(reduce_tuning<target_block_size>{}, scan_tuning{});

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Reduce(d_in, d_out.begin(), num_items, block_size_check, 0, env));
  REQUIRE(d_out[0] == num_items);
  REQUIRE(d_block_size[0] == target_block_size);
}
#endif

using requirements =
  c2h::type_list<cuda::execution::determinism::run_to_run_t, cuda::execution::determinism::not_guaranteed_t>;

C2H_TEST("Device reduce uses environment", "[reduce][device]", requirements)
{
  using determinism_t = c2h::get<0, TestType>;
  using accumulator_t = int;
  using op_t          = cuda::std::plus<>;
  using num_items_t   = int;
  using offset_t      = hipcub::detail::choose_offset_t<num_items_t>;
  using transform_t   = ::cuda::std::__identity;
  using policy_t      = hipcub::detail::reduce::policy_hub<accumulator_t, offset_t, op_t>::MaxPolicy;
  using init_t        = accumulator_t;

  num_items_t num_items = GENERATE(1 << 4, 1 << 24);
  auto d_in             = thrust::make_constant_iterator(1);
  auto d_out            = thrust::device_vector<accumulator_t>(1);

  // To check if a given algorithm implementation is used, we check if associated kernels are invoked.
  auto kernels = [&]() {
    // TODO(gevtushenko): split `not_guaranteed` kernels once atomic reduce is merged
    if constexpr (std::is_same_v<determinism_t, cuda::execution::determinism::run_to_run_t>
                  || std::is_same_v<determinism_t, cuda::execution::determinism::not_guaranteed_t>)
    {
      return cuda::std::array<void*, 3>{
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            decltype(d_in),
            decltype(d_out.begin()),
            offset_t,
            op_t,
            init_t,
            accumulator_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceKernel<policy_t, decltype(d_in), offset_t, op_t, accumulator_t, transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            accumulator_t*,
            decltype(d_out.begin()),
            int, // always used with int offset
            op_t,
            init_t,
            accumulator_t>)};
    }
    else
    {
      // TODO(gevtushenko): add `gpu_to_gpu` kernels once RFA is merged
      FAIL("Only run_to_run and not_guaranteed determinism are supported at the moment");
      return cuda::std::array<void*, 0>{};
    }
  }();

  init_t init = 0;

  size_t expected_bytes_allocated{};
  REQUIRE(hipSuccess
          == hipcub::DeviceReduce::Reduce(
            nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items, cuda::std::plus<>{}, init));

  // Equivalent to `cuexec::require(cuexec::determinism::run_to_run)` and
  //               `cuexec::require(cuexec::determinism::not_guaranteed)`
  auto env = stdexec::env{cuda::execution::require(determinism_t{}), // determinism
                          allowed_kernels(kernels), // allowed kernels for the given determinism
                          expected_allocation_size(expected_bytes_allocated)}; // temp storage size

  // TODO(gevtushenko): how to check if given requirement is met?
  device_reduce(d_in, d_out.begin(), num_items, cuda::std::plus<>{}, init, env);

  REQUIRE(d_out[0] == num_items);
}
