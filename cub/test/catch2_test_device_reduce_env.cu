#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

struct stream_registry_factory_t;
#define CUB_DETAIL_DEFAULT_KERNEL_LAUNCHER stream_registry_factory_t

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>

// #include <cuda/experimental/__execution/env.cuh>
// #include <cuda/experimental/memory_resource.cuh>
// #include <cuda/experimental/stream.cuh>

#include <cuda/std/optional>

#include <cstdint>

#include <c2h/catch2_test_helper.h>

struct stream_registry_factory_t
{
  cuda::std::optional<hipStream_t> m_stream;

  thrust::cuda_cub::detail::triple_chevron
  operator()(dim3 grid, dim3 block, size_t shared_mem, hipStream_t stream, bool dependent_launch = false) const
  {
    if (m_stream)
    {
      REQUIRE(stream == m_stream);
    }
    return thrust::cuda_cub::detail::triple_chevron(grid, block, shared_mem, stream, dependent_launch);
  }

  hipError_t PtxVersion(int& version)
  {
    return hipcub::PtxVersion(version);
  }

  hipError_t MultiProcessorCount(int& sm_count) const
  {
    int device_ordinal;
    hipError_t error = hipGetDevice(&device_ordinal);
    if (hipSuccess != error)
    {
      return error;
    }

    // Get SM count
    return hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal);
  }

  template <typename Kernel>
  hipError_t MaxSmOccupancy(int& sm_occupancy, Kernel kernel_ptr, int block_size, int dynamic_smem_bytes = 0)
  {
    return hipOccupancyMaxActiveBlocksPerMultiprocessor(&sm_occupancy, kernel_ptr, block_size, dynamic_smem_bytes);
  }

  hipError_t MaxGridDimX(int& max_grid_dim_x) const
  {
    int device_ordinal;
    hipError_t error = hipGetDevice(&device_ordinal);
    if (hipSuccess != error)
    {
      return error;
    }

    // Get max grid dimension
    return hipDeviceGetAttribute(&max_grid_dim_x, hipDeviceAttributeMaxGridDimX, device_ordinal);
  }
};

// singleton
stream_registry_factory_t& get_stream_registry_factory()
{
  static stream_registry_factory_t factory;
  return factory;
}

struct stream_scope
{
  stream_scope(hipStream_t stream)
  {
    get_stream_registry_factory().m_stream = stream;
  }

  ~stream_scope()
  {
    get_stream_registry_factory().m_stream = cuda::std::nullopt;
  }
};

TEST_CASE("Device reduce works with default environment", "[reduce][device]")
{
  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  hipError_t err = hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0);
  REQUIRE(err == hipSuccess);

  REQUIRE(d_out[0] == 15);
}

TEST_CASE("Device reduce works with cudax environment", "[reduce][device]")
{
  hipStream_t stream;
  REQUIRE(hipStreamCreate(&stream) == hipSuccess);

  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  cuda::std::execution::prop env{cuda::get_stream, stream};

  {
    stream_scope scope(stream);
    REQUIRE(
      hipSuccess == hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0, env));
  }

  REQUIRE(d_out[0] == 15);

  REQUIRE(hipStreamDestroy(stream) == hipSuccess);
}
