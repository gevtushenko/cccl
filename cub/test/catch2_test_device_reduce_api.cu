#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

struct StreamRegistryFactory;
#define CUB_DETAIL_DEFAULT_KERNEL_LAUNCHER StreamRegistryFactory

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>

#include <cuda/experimental/__execution/env.cuh>
#include <cuda/experimental/memory_resource.cuh>
#include <cuda/experimental/stream.cuh>

#include <cstdint>

#include <c2h/catch2_test_helper.h>

struct StreamRegistryFactory
{
  thrust::cuda_cub::detail::triple_chevron
  operator()(dim3 grid, dim3 block, size_t shared_mem, hipStream_t stream, bool dependent_launch = false) const
  {
    std::cout << "Launch on " << stream << std::endl;
    return thrust::cuda_cub::detail::triple_chevron(grid, block, shared_mem, stream, dependent_launch);
  }

  hipError_t PtxVersion(int& version)
  {
    return hipcub::PtxVersion(version);
  }

  hipError_t MultiProcessorCount(int& sm_count) const
  {
    int device_ordinal;
    hipError_t error = hipGetDevice(&device_ordinal);
    if (hipSuccess != error)
    {
      return error;
    }

    // Get SM count
    return hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal);
  }

  template <typename Kernel>
  hipError_t MaxSmOccupancy(int& sm_occupancy, Kernel kernel_ptr, int block_size, int dynamic_smem_bytes = 0)
  {
    return hipOccupancyMaxActiveBlocksPerMultiprocessor(&sm_occupancy, kernel_ptr, block_size, dynamic_smem_bytes);
  }

  hipError_t MaxGridDimX(int& max_grid_dim_x) const
  {
    int device_ordinal;
    hipError_t error = hipGetDevice(&device_ordinal);
    if (hipSuccess != error)
    {
      return error;
    }

    // Get max grid dimension
    return hipDeviceGetAttribute(&max_grid_dim_x, hipDeviceAttributeMaxGridDimX, device_ordinal);
  }
};

namespace cudax = cuda::experimental;

TEST_CASE("Device reduce works with default environment", "[reduce][device]")
{
  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  hipError_t err = hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0);
  REQUIRE(err == hipSuccess);

  REQUIRE(d_out[0] == 15);
}

TEST_CASE("Device reduce works with cudax environment", "[reduce][device]")
{
  cudax::stream stream;
  cudax::env_t<cuda::mr::device_accessible> env{cudax::device_memory_resource{}, stream};

  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  hipError_t err = hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0, env);
  REQUIRE(err == hipSuccess);

  REQUIRE(d_out[0] == 15);
}
