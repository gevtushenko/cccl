/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>

#include <cuda/experimental/__execution/env.cuh>
#include <cuda/experimental/memory_resource.cuh>
#include <cuda/experimental/stream.cuh>

#include <cstdint>

#include <c2h/catch2_test_helper.h>

namespace cudax = cuda::experimental;

TEST_CASE("Device reduce works with default environment", "[reduce][device]")
{
  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  hipError_t err = hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0);
  REQUIRE(err == hipSuccess);

  REQUIRE(d_out[0] == 15);
}

TEST_CASE("Device reduce works with cudax environment", "[reduce][device]")
{
  cudax::stream stream;
  cudax::env_t<cuda::mr::device_accessible> env{cudax::device_memory_resource{}, stream};

  thrust::device_vector<int> d_in{1, 2, 3, 4, 5};
  thrust::device_vector<int> d_out(1);

  hipError_t err = hipcub::DeviceReduce::Reduce(d_in.begin(), d_out.begin(), d_in.size(), cuda::std::plus<>{}, 0, env);
  REQUIRE(err == hipSuccess);

  REQUIRE(d_out[0] == 15);
}
